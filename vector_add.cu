#include <stdio.h>
#include <hip/hip_runtime.h>

// CUDA Kernel for adding numbers
__global__ void addNumbers(int *input, int *result) {
    int tid = threadIdx.x;
    
    // Each thread adds one number to the result
    atomicAdd(result, input[tid]);
}

int main() {
    // Input array with 5 numbers
    int h_input[5] = {1, 1, 1, 1, 1};
    int h_result = 0;
    
    // Allocate device memory
    int *d_input, *d_result;
    hipMalloc(&d_input, 5 * sizeof(int));
    hipMalloc(&d_result, sizeof(int));
    
    // Copy input array to device
    hipMemcpy(d_input, h_input, 5 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_result, &h_result, sizeof(int), hipMemcpyHostToDevice);
    
    // Launch kernel with 5 threads in one block
    addNumbers<<<1, 5>>>(d_input, d_result);
    
    // Copy result back to host
    hipMemcpy(&h_result, d_result, sizeof(int), hipMemcpyDeviceToHost);
    
    // Print result
    printf("Sum of numbers: %d\n", h_result);
    
    // Cleanup
    hipFree(d_input);
    hipFree(d_result);
    
    return 0;
} 